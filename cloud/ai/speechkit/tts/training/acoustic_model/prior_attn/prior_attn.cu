#include "hip/hip_runtime.h"
#include <torch/extension.h>

template <typename T>
__device__ T betaln(T a, T b) {
    return lgammaf(a) + lgammaf(b) - lgammaf(a + b);
}

template <typename T>
__device__ T pmf(T x, T n, T a, T b) {
    const auto combiln = -logf(n + 1) - betaln(n - x + 1, x + 1);
    return exp(combiln + betaln(x + a, n - x + b) - betaln(a, b));
}

template<class T>
__global__ void prior_attn_kernel(torch::PackedTensorAccessor32<T, 3, torch::RestrictPtrTraits> array,
                                  torch::PackedTensorAccessor32<int64_t, 1, torch::RestrictPtrTraits> m_lengths,
                                  torch::PackedTensorAccessor32<int64_t, 1, torch::RestrictPtrTraits> n_lengths) {
    const auto n = blockIdx.z; // batch idx
    const auto i = threadIdx.x + blockIdx.x * blockDim.x;
    const auto j = threadIdx.y + blockIdx.y * blockDim.y;

    const auto M = m_lengths[n];
    const auto N = n_lengths[n];

    if (i < M && j < N && i < array.size(1) && j < array.size(2)) {
        array[n][i][j] = pmf(T(j), T(N), T(i + 1), T(M - i));
    }
}

void prior_attn(torch::Tensor array, torch::Tensor m_lengths, torch::Tensor n_lengths) {
    AT_DISPATCH_FLOATING_TYPES(
        array.scalar_type(), "prior_attn", [&]() {
            const int64_t threads = 32;
            const int64_t batch_size = array.size(0);
            const int64_t M = array.size(1);
            const int64_t N = array.size(2);
            const int64_t n_blocks_x = (M % threads) ? M / threads + 1 : M / threads;
            const int64_t n_blocks_y = (N % threads) ? N / threads + 1 : N / threads;
            prior_attn_kernel<<<dim3(n_blocks_x, n_blocks_y, batch_size), dim3(threads, threads)>>>(
                array.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>(),
                m_lengths.packed_accessor32<int64_t, 1, torch::RestrictPtrTraits>(),
                n_lengths.packed_accessor32<int64_t, 1, torch::RestrictPtrTraits>()
            );
        }
    );
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("prior_attn", &prior_attn, "generate prior attention matrix");
}
